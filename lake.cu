
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#include <time.h>

#define XMIN 0.0
#define XMAX 1.0
#define YMIN 0.0
#define YMAX 1.0

#define MAX_PSZ 10
#define TSCALE 1.0
#define VSQR 0.1

void init(double *u, double *pebbles, int n);
void evolve(double *un, double *uc, double *uo, double *pebbles, int n,
            double h, double dt, double t);
extern int tpdt(double *t, double dt, double end_time);
void print_heatmap(const char *filename, double *u, int n, double h);
void init_pebbles(double *p, int pn, int n);
void run_cpu(double *u, double *u0, double *u1, double *pebbles, int n,
             double h, double end_time);
extern void run_gpu(double *u, double *u0, double *u1, double *pebbles, int n,
                    double h, double end_time, int nthreads);

int main(int argc, char *argv[]) {

  if (argc != 5) {
    printf("Usage: %s npoints npebs time_finish nthreads \n", argv[0]);
    return 0;
  }

  int npoints = atoi(argv[1]);
  int npebs = atoi(argv[2]);
  double end_time = (double)atof(argv[3]);
  int nthreads = atoi(argv[4]);
  int narea = npoints * npoints;

  double *u_i0, *u_i1;
  double *u_cpu, *u_gpu, *pebs;
  double h;

  double elapsed_cpu, elapsed_gpu;
  struct timeval cpu_start, cpu_end, gpu_start, gpu_end;

  u_i0 = (double *)malloc(sizeof(double) * narea);
  u_i1 = (double *)malloc(sizeof(double) * narea);
  pebs = (double *)malloc(sizeof(double) * narea);

  u_cpu = (double *)malloc(sizeof(double) * narea);
  u_gpu = (double *)malloc(sizeof(double) * narea);

  printf("Running %s with (%d x %d) grid, until %f, with %d threads\n", argv[0],
         npoints, npoints, end_time, nthreads);

  h = (XMAX - XMIN) / npoints;

  init_pebbles(pebs, npebs, npoints);
  init(u_i0, pebs, npoints);
  init(u_i1, pebs, npoints);

  print_heatmap("lake_i_cuda.dat", u_i0, npoints, h);

  gettimeofday(&cpu_start, NULL);
  run_cpu(u_cpu, u_i0, u_i1, pebs, npoints, h, end_time);
  gettimeofday(&cpu_end, NULL);

  elapsed_cpu = ((cpu_end.tv_sec + cpu_end.tv_usec * 1e-6) -
                 (cpu_start.tv_sec + cpu_start.tv_usec * 1e-6));
  printf("CPU took %f seconds\n", elapsed_cpu);

  gettimeofday(&gpu_start, NULL);
  run_gpu(u_gpu, u_i0, u_i1, pebs, npoints, h, end_time, nthreads);
  gettimeofday(&gpu_end, NULL);
  elapsed_gpu = ((gpu_end.tv_sec + gpu_end.tv_usec * 1e-6) -
                 (gpu_start.tv_sec + gpu_start.tv_usec * 1e-6));
  printf("GPU took %f seconds\n", elapsed_gpu);

  print_heatmap("lake_f_cuda.dat", u_cpu, npoints, h);
  print_heatmap("lake_f_gpu_cuda.dat", u_gpu, npoints, h);

  free(u_i0);
  free(u_i1);
  free(pebs);
  free(u_cpu);
  free(u_gpu);

  return 0;
}

void run_cpu(double *u, double *u0, double *u1, double *pebbles, int n,
             double h, double end_time) {
  double *un, *uc, *uo, *temp;
  double t, dt;

  un = (double *)malloc(sizeof(double) * n * n);
  uc = (double *)malloc(sizeof(double) * n * n);
  uo = (double *)malloc(sizeof(double) * n * n);

  memcpy(uo, u0, sizeof(double) * n * n);
  memcpy(uc, u1, sizeof(double) * n * n);

  t = 0.;
  dt = h / 2.;

  while (1) {
    evolve(un, uc, uo, pebbles, n, h, dt, t);

    temp = uo;
    uo = uc;
    uc = un;
    un = temp;

    if (!tpdt(&t, dt, end_time))
      break;
  }

  memcpy(u, un, sizeof(double) * n * n);
  free(uo);
  free(uc);
  free(un);
}

void init_pebbles(double *p, int pn, int n) {
  int i, j, k, idx;
  int sz;

  srand(time(NULL));
  memset(p, 0, sizeof(double) * n * n);

  p[64 * 256 + 64] = 2;
  p[64 * 256 + 192] = 2;
  p[192 * 256 + 64] = 2;
  p[192 * 256 + 192] = 2;

  // for (k = 0; k < pn; k++) {
  //   i = rand() % (n - 4) + 2;
  //   j = rand() % (n - 4) + 2;
  //   sz = rand() % MAX_PSZ;
  //   idx = j + i * n;
  //   p[idx] = (double)sz;
  // }
}

double f(double p, double t) { return -expf(-TSCALE * t) * p; }

void init(double *u, double *pebbles, int n) {
  int i, j, idx;

  for (i = 0; i < n; i++) {
    for (j = 0; j < n; j++) {
      idx = j + i * n;
      u[idx] = f(pebbles[idx], 0.0);
    }
  }
}

void evolve(double *un, double *uc, double *uo, double *pebbles, int n,
            double h, double dt, double t) {
  int i, j, idx;

  for (i = 0; i < n; i++) {
    for (j = 0; j < n; j++) {
      idx = j + i * n;
      // boundary condition check
      if (i == 0 || i == 1 || i == n - 1 || i == n - 2 || j == 0 || j == 1 ||
          j == n - 1 || j == n - 2) {
        un[idx] = 0.;
      } else {

        // goal calculation
        // un[idx] = 2*uc[idx] - uo[idx] + VSQR *(dt * dt) *(( WEST + EAST +
        // NORTH + SOUTH + 0.25*(NORTHWEST + NORTHEAST + SOUTHWEST +
        // SOUTHEAST)
        // + 0.125*(WESTWEST + EASTEAST + NORTHNORTH + SOUTHSOUTH) - 5.5 *
        // uc[idx])/(h * h) + f(pebbles[idx],t));

        un[idx] =
            2 * uc[idx] - uo[idx] +
            VSQR * (dt * dt) *
                ((1 * (uc[idx - 1] + uc[idx + 1] + uc[idx + n] +
                       uc[idx - n]) + // 1st degree cardinals
                  0.25 * (uc[idx + n - 1] + uc[idx + n + 1] + uc[idx - n - 1] +
                          uc[idx - n + 1]) + // 1st degree ordinals
                  0.125 * (uc[idx - 2] + uc[idx + 2] + uc[idx + n + n] +
                           uc[idx - n - n]) - // 2nd degree cardinals
                  5.5 * uc[idx]) /
                     (h * h) + // normalization
                 f(pebbles[idx], t));
      }
    }
  }
}

void print_heatmap(const char *filename, double *u, int n, double h) {
  int i, j, idx;

  FILE *fp = fopen(filename, "w");

  for (i = 0; i < n; i++) {
    for (j = 0; j < n; j++) {
      idx = j + i * n;
      fprintf(fp, "%f %f %f\n", i * h, j * h, u[idx]);
    }
  }

  fclose(fp);
}
